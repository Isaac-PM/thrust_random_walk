#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>
#include <filesystem>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <hiprand/hiprand_kernel.h>
#include <vtkNew.h>
#include <vtkPoints.h>
#include <vtkPolyData.h>
#include <vtkCellArray.h>
#include <vtkPolyLine.h>
#include <vtkXMLPolyDataWriter.h>

#define EXPECTED_ARG_COUNT 3 // <program_name> <particle_count> <steps>

__host__ void printUsage(const std::string &programName);

#define PI 3.14159265358979323846

#define THREAD_COUNT_1D 1024

#define DEFAULT_PARTICLE_COUNT 1'000
#define DEFAULT_STEPS 2'000
#define STEP_SIZE 0.001 // Also known as R

#define WRITE_FREQUENCY 2
#define OUTPUT_DIR "output"

struct Particle
{
    double x, y, z;
};

struct FunctorInitParticles
{
    Particle *particles;

    __host__ __device__ FunctorInitParticles(Particle *particles) : particles(particles) {}

    __device__ void operator()(const size_t idx) const
    {
        particles[idx].x = 0.0;
        particles[idx].y = 0.0;
        particles[idx].z = 0.0;
    }
};

struct FunctorRandomWalk
{
    Particle *positions;
    hiprandState *states;

    __host__ __device__ FunctorRandomWalk(Particle *positions, hiprandState *states) : positions(positions), states(states) {}

    __device__ void operator()(const size_t idx) const
    {
        hiprandState localState = states[idx];

        double theta = PI * hiprand_uniform(&localState);
        double phi = 2.0 * PI * hiprand_uniform(&localState);

        double dx = STEP_SIZE * sin(theta) * cos(phi);
        double dy = STEP_SIZE * sin(theta) * sin(phi);
        double dz = STEP_SIZE * cos(theta);

        positions[idx].x += dx;
        positions[idx].y += dy;
        positions[idx].z += dz;

        states[idx] = localState;
    }
};

__global__ void initCurandStates(hiprandState *states, const size_t stateCount, const size_t seed)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < stateCount)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

int main(int argc, char *argv[])
{
    size_t particleCount = DEFAULT_PARTICLE_COUNT;
    size_t steps = DEFAULT_STEPS;

    if (argc == EXPECTED_ARG_COUNT)
    {
        particleCount = std::stoul(argv[1]);
        steps = std::stoul(argv[2]);
    }
    else
    {
        printUsage(argv[0]);
        std::cout << "Using default values: " << particleCount << " particles, " << steps << " steps\n";
    }

    if (!std::filesystem::exists(OUTPUT_DIR))
    {
        std::filesystem::create_directory(OUTPUT_DIR);
    }

    thrust::device_vector<Particle> d_particles(particleCount);
    thrust::for_each(thrust::counting_iterator<size_t>(0),
                     thrust::counting_iterator<size_t>(particleCount),
                     FunctorInitParticles(thrust::raw_pointer_cast(d_particles.data())));

    hiprandState *d_states;
    hipMalloc(&d_states, particleCount * sizeof(hiprandState));
    dim3 dimBlock(THREAD_COUNT_1D, 1, 1);
    dim3 dimGrid((particleCount + THREAD_COUNT_1D - 1) / THREAD_COUNT_1D, 1, 1);
    initCurandStates<<<dimGrid, dimBlock>>>(d_states, particleCount, 0);
    hipDeviceSynchronize();

    for (size_t step = 0; step < steps; ++step)
    {
        if (step % WRITE_FREQUENCY == 0)
        {
            thrust::host_vector<Particle> h_particles = d_particles;
            hipDeviceSynchronize();
            vtkNew<vtkPoints> points;
            for (size_t i = 0; i < particleCount; ++i)
            {
                points->InsertNextPoint(h_particles[i].x, h_particles[i].y, h_particles[i].z);
            }
            std::stringstream filename;
            filename << OUTPUT_DIR << "/random_walk_step_" << step << ".vtp";
            vtkNew<vtkPolyData> polyData;
            polyData->SetPoints(points);
            vtkNew<vtkXMLPolyDataWriter> writer;
            writer->SetFileName(filename.str().c_str());
            writer->SetInputData(polyData);
            writer->Write();
        }
        thrust::for_each(thrust::counting_iterator<int>(0),
                         thrust::counting_iterator<int>(particleCount),
                         FunctorRandomWalk(thrust::raw_pointer_cast(d_particles.data()), d_states));
    }

    hipFree(d_states);

    return EXIT_SUCCESS;
}

__host__ void printUsage(const std::string &programName)
{
    std::cout << "Usage: " << programName << " <particle_count> <steps>\n";
    std::cout << "   <particle_count> - number of particles to simulate\n";
    std::cout << "   <steps>          - number of simulation steps to perform\n";
}